#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------------------------------
// Copyrighted by Marko Rakita.
// Author: Marko Rakita
// File contains: Neural network standard layer.
// Created: 01/17/2016.
// ----------------------------------------------------------------------------------------------------

#include "include/standardlayer.cuh"

StandardLayer::StandardLayer(ParallelismMode parallelismMode, hipStream_t deviceCalculationStream, hipStream_t deviceMemoryStream, hipblasHandle_t cublasHandle,
	uint indexInTier, uint tierSize, uint inputNumChannels, uint inputDataWidth, uint inputDataHeight, uint inputDataCount, bool holdsInputData, uint numNeurons, bool initializeWeights,
	float weightsDeviation, bool initializeBiases, float biasesInitialValue, float weightsUpdateMomentum, float weightsUpdateDecay, float weightsUpdateLearningRateProgressStep,
	float weightsUpdateStartingLearningRate, float weightsUpdateLearningRateUpdateFactor, float biasesUpdateMomentum, float biasesUpdateDecay, float biasesUpdateLearningRateProgressStep,
	float biasesUpdateStartingLearningRate, float biasesUpdateLearningRateUpdateFactor, ActivationType activationType, bool holdsActivationGradients)
{
	m_layerType = LayerType::Standard;
	m_parallelismMode = parallelismMode;
	m_deviceCalculationStream = deviceCalculationStream;
	m_deviceMemoryStream = deviceMemoryStream;
	m_cublasHandle = cublasHandle;
	m_indexInTier = indexInTier;
	m_tierSize = tierSize;
	m_activationType = activationType;

	m_inputNumChannels = inputNumChannels;
	m_inputDataWidth = inputDataWidth;
	m_inputDataHeight = inputDataHeight;
	m_inputDataSize = m_inputDataWidth * m_inputDataHeight;
	m_inputDataCount = inputDataCount;
	m_holdsInputData = holdsInputData;

	m_numNeurons = numNeurons;
	m_numWeightsPerNeuron = m_inputNumChannels * m_inputDataSize;

	m_weightsUpdateMomentum = weightsUpdateMomentum;
	m_weightsUpdateDecay = weightsUpdateDecay;
	m_weightsUpdateLearningRateProgressStep = weightsUpdateLearningRateProgressStep;
	m_weightsUpdateStartingLearningRate = weightsUpdateStartingLearningRate;
	m_weightsUpdateLearningRateUpdateFactor = weightsUpdateLearningRateUpdateFactor;

	m_biasesUpdateMomentum = biasesUpdateMomentum;
	m_biasesUpdateDecay = biasesUpdateDecay;
	m_biasesUpdateLearningRateProgressStep = biasesUpdateLearningRateProgressStep;
	m_biasesUpdateStartingLearningRate = biasesUpdateStartingLearningRate;
	m_biasesUpdateLearningRateUpdateFactor = biasesUpdateLearningRateUpdateFactor;

	m_activationNumChannels = 1;
	m_activationDataWidth = m_numNeurons;
	m_activationDataHeight = 1;
	m_activationDataSize = m_activationDataWidth * m_activationDataHeight;

	// Allocating input data buffer.
	m_inputBufferSize = m_inputNumChannels * m_inputDataSize * m_inputDataCount * sizeof(float);
	if (m_holdsInputData)
	{
		CudaAssert(hipMalloc<float>(&m_inputDataBuffer, m_inputBufferSize));
	}

	// Allocating input gradients buffer.
	CudaAssert(hipMalloc<float>(&m_inputGradientsBuffer, m_inputBufferSize));

	// Allocating weights buffers.
	m_weightsBufferSize = m_numNeurons * m_numWeightsPerNeuron * sizeof(float);
	CudaAssert(hipMalloc<float>(&m_weightsBuffer, m_weightsBufferSize));
	CudaAssert(hipMalloc<float>(&m_weightsGradientsBuffer, m_weightsBufferSize));
	CudaAssert(hipMalloc<float>(&m_weightsUpdateBuffer, m_weightsBufferSize));

	// Initializing weights.
	if (initializeWeights)
	{
		InitializeParamsFromDistribution(m_weightsBuffer, m_weightsBufferSize, weightsDeviation);
		InitializeParamsToValue(m_weightsUpdateBuffer, m_weightsBufferSize, 0.f);
	}

	// Allocating biases buffer.
	m_biasesBufferSize = m_numNeurons * sizeof(float);
	CudaAssert(hipMalloc<float>(&m_biasesBuffer, m_biasesBufferSize));
	CudaAssert(hipMalloc<float>(&m_biasesGradientsBuffer, m_biasesBufferSize));
	CudaAssert(hipMalloc<float>(&m_biasesUpdateBuffer, m_biasesBufferSize));

	// Initializing biases.
	if (initializeBiases)
	{
		InitializeParamsToValue(m_biasesBuffer, m_biasesBufferSize, biasesInitialValue);
		InitializeParamsToValue(m_biasesUpdateBuffer, m_biasesBufferSize, 0.f);
	}

	// Allocating preactivation and activation data buffers.
	m_activationBufferSize = m_inputDataCount * m_activationDataSize * sizeof(float);
	CudaAssert(hipMalloc<float>(&m_preactivationDataBuffer, m_activationBufferSize));
	CudaAssert(hipMalloc<float>(&m_activationDataBuffer, m_activationBufferSize));

	// Allocating preactivation gradients buffer.
	CudaAssert(hipMalloc<float>(&m_preactivationGradientsBuffer, m_activationBufferSize));

	// Allocating activation gradients buffer.
	m_holdsActivationGradients = holdsActivationGradients;
	if (m_holdsActivationGradients)
	{
		CudaAssert(hipMalloc<float>(&m_activationGradientsBuffer, m_activationBufferSize));
	}
}

void StandardLayer::Reinitialize(uint newInputDataCount)
{
	Layer::Reinitialize(newInputDataCount);

	m_activationBufferSize = m_inputDataCount * m_activationDataSize * sizeof(float);
}

void StandardLayer::CopyWeightsFromHost(float* hostWeightsBuffer)
{
	CudaAssert(hipMemcpyAsync(m_weightsBuffer, hostWeightsBuffer, m_weightsBufferSize, hipMemcpyHostToDevice, m_deviceMemoryStream));
	SynchronizeMemoryOperations();
}

void StandardLayer::CopyWeightsUpdateFromHost(float* hostWeightsUpdateBuffer)
{
	CudaAssert(hipMemcpyAsync(m_weightsUpdateBuffer, hostWeightsUpdateBuffer, m_weightsBufferSize, hipMemcpyHostToDevice, m_deviceMemoryStream));
	SynchronizeMemoryOperations();
}

void StandardLayer::CopyBiasesFromHost(float* hostBiasesBuffer)
{
	CudaAssert(hipMemcpyAsync(m_biasesBuffer, hostBiasesBuffer, m_biasesBufferSize, hipMemcpyHostToDevice, m_deviceMemoryStream));
	SynchronizeMemoryOperations();
}

void StandardLayer::CopyBiasesUpdateFromHost(float* hostBiasesUpdateBuffer)
{
	CudaAssert(hipMemcpyAsync(m_biasesUpdateBuffer, hostBiasesUpdateBuffer, m_biasesBufferSize, hipMemcpyHostToDevice, m_deviceMemoryStream));
	SynchronizeMemoryOperations();
}

StandardLayer::~StandardLayer()
{
	CudaAssert(hipFree(m_weightsBuffer));
	CudaAssert(hipFree(m_weightsGradientsBuffer));
	CudaAssert(hipFree(m_weightsUpdateBuffer));

	CudaAssert(hipFree(m_biasesBuffer));
	CudaAssert(hipFree(m_biasesGradientsBuffer));
	CudaAssert(hipFree(m_biasesUpdateBuffer));

	CudaAssert(hipFree(m_preactivationDataBuffer));
	CudaAssert(hipFree(m_preactivationGradientsBuffer));
}

void StandardLayer::LoadInputs()
{
	CommonLoadInputs();
}

void StandardLayer::CalculatePreactivations()
{
	CudaCublasAssert(hipblasSetStream(m_cublasHandle, m_deviceCalculationStream));
	float alpha = 1.0f;
	float beta = 0.f;
	CudaCublasAssert(hipblasSgemm(m_cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, (int)m_inputDataCount, (int)m_numNeurons, (int)m_numWeightsPerNeuron,
		&alpha, m_inputDataBuffer, (int)m_inputDataCount, m_weightsBuffer, (int)m_numWeightsPerNeuron, &beta, m_preactivationDataBuffer, (int)m_inputDataCount));
}

/*
	Does grid stride and adds biases to preactivations.
*/
__global__ void AddNeuronBiases(float* preactivations, float* biases, const uint width, const uint height)
{
	for (uint y = blockIdx.y; y < height; y += gridDim.y)
	{
		int laneId = threadIdx.x % warpSize;
		int biasValue;
		if (laneId == 0)
		{
			biasValue = biases[y];
		}
		biasValue = __shfl(biasValue, 0);

		for (uint x = blockIdx.x * blockDim.x + threadIdx.x; x < width; x += gridDim.x * blockDim.x)
		{
			preactivations[y * width + x] += biasValue;
		}
	}
}

void StandardLayer::AddBiases()
{
	const uint c_numThreadsPerBlock = min((uint)Config::MAX_NUM_THREADS, RoundUp(m_inputDataCount, Config::WARP_SIZE));
	dim3 blockDimensions(c_numThreadsPerBlock);
	uint c_numBlocks = (uint)((Config::MAX_NUM_THREADS / c_numThreadsPerBlock) * 128);
	dim3 gridDimensions(1, c_numBlocks);
	LAUNCH_KERNEL_ASYNC(AddNeuronBiases, gridDimensions, blockDimensions, m_deviceCalculationStream)(m_preactivationDataBuffer, m_biasesBuffer,
		m_inputDataCount, m_numNeurons);
	CudaAssert(hipGetLastError());
}

void StandardLayer::CalculateActivations()
{
	ApplyActivation(m_activationType, m_preactivationDataBuffer, (uint)(m_activationBufferSize / sizeof(float)), m_activationDataBuffer, m_deviceCalculationStream);
}

void StandardLayer::DoForwardProp(PropagationMode propagationMode)
{
	CalculatePreactivations();
	AddBiases();
	CalculateActivations();
}

/*
	Calculates biases gradients, each thread calculating gradient for one bias.
*/
__global__ void __CalculateStandardBiasesGradients(float* preactivationGradients, const uint numNeurons, const uint inputDataCount, const uint batchSize,
	float* biasesGradients)
{
	const uint c_neuronIndex = blockIdx.x * blockDim.x + threadIdx.x;
	const uint c_neuronPreactivationsOffset = c_neuronIndex * inputDataCount;

	if (c_neuronIndex < numNeurons)
	{
		float biasGradient = 0.f;
		for (uint dataIndex = 0; dataIndex < inputDataCount; ++dataIndex)
		{
			biasGradient += preactivationGradients[c_neuronPreactivationsOffset + dataIndex];
		}

		biasesGradients[c_neuronIndex] = biasGradient / (float)batchSize;
	}
}

void StandardLayer::CalculateBiasesGradients()
{
	const uint c_numThreadsPerBlock = 128;
	const uint c_numBlocks = DivideUp(m_numNeurons, c_numThreadsPerBlock);
	const uint c_batchSize = m_parallelismMode == ParallelismMode::Model ? m_inputDataCount : m_tierSize * m_inputDataCount;
	LAUNCH_KERNEL_ASYNC(__CalculateStandardBiasesGradients, dim3(c_numBlocks), dim3(c_numThreadsPerBlock), m_deviceCalculationStream)(m_preactivationGradientsBuffer,
		m_numNeurons, m_inputDataCount, c_batchSize, m_biasesGradientsBuffer);
	CudaAssert(hipGetLastError());
}

void StandardLayer::CalculateWeightsGradients()
{
	CudaCublasAssert(hipblasSetStream(m_cublasHandle, m_deviceCalculationStream));
	uint batchSize = m_parallelismMode == ParallelismMode::Model ? m_inputDataCount : m_tierSize * m_inputDataCount;
	float alpha = 1.0f / batchSize;
	float beta = 0.f;
	CudaCublasAssert(hipblasSgemm(m_cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, (int)m_numWeightsPerNeuron, (int)m_numNeurons, (int)m_inputDataCount,
		&alpha, m_inputDataBuffer, (int)m_inputDataCount, m_preactivationGradientsBuffer, (int)m_inputDataCount, &beta, m_weightsGradientsBuffer, (int)m_numWeightsPerNeuron));
}

void StandardLayer::CalculateInputGradients()
{
	CudaCublasAssert(hipblasSetStream(m_cublasHandle, m_deviceCalculationStream));
	float alpha = 1.0f;
	float beta = 0.f;
	CudaCublasAssert(hipblasSgemm(m_cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, (int)m_inputDataCount, (int)m_numWeightsPerNeuron, (int)m_numNeurons,
		&alpha, m_preactivationGradientsBuffer, (int)m_inputDataCount, m_weightsBuffer, (int)m_numWeightsPerNeuron, &beta, m_inputGradientsBuffer, (int)m_inputDataCount));
}

void StandardLayer::CalculatePreactivationsGradients()
{
	CalculatePreactivationGradients(m_activationType, m_activationGradientsBuffer, m_activationDataBuffer, (uint)(m_activationBufferSize / sizeof(float)),
		m_preactivationGradientsBuffer, m_deviceCalculationStream);
}

void StandardLayer::DoBackwardProp()
{
	CalculatePreactivationsGradients();
	CalculateInputGradients();
	CalculateWeightsGradients();
	CalculateBiasesGradients();
}

void StandardLayer::UpdateLayerParameters(float learningProgress)
{
	CommonUpdateLayerParameters(learningProgress, m_weightsBuffer, m_weightsGradientsBuffer, m_weightsUpdateBuffer, (uint)(m_weightsBufferSize / sizeof(float)),
		m_weightsUpdateMomentum, m_weightsUpdateLearningRateProgressStep, m_weightsUpdateStartingLearningRate, m_weightsUpdateLearningRateUpdateFactor,
		m_weightsUpdateDecay, m_biasesBuffer, m_biasesGradientsBuffer, m_biasesUpdateBuffer, (uint)(m_biasesBufferSize / sizeof(float)), m_biasesUpdateMomentum,
		m_biasesUpdateLearningRateProgressStep, m_biasesUpdateStartingLearningRate, m_biasesUpdateLearningRateUpdateFactor, m_biasesUpdateDecay);
}