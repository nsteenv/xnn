#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------------------------------
// Copyrighted by Marko Rakita.
// Author: Marko Rakita
// File contains: Prepares data for training.
// Created: 11/24/2015.
// ----------------------------------------------------------------------------------------------------

#include "include/datamaker.cuh"

const string DataMaker::c_inputFolderSignature = "-inputfolder";
const string DataMaker::c_inputDataListSignature = "-inputdatalist";
const string DataMaker::c_outputFolderSignature = "-outputfolder";
const string DataMaker::c_imageSizeSignature = "-imagesize";
const string DataMaker::c_numImageChannelsSignature = "-numchannels";

const uint DataMaker::c_defaultImageSize = 256;
const uint DataMaker::c_defaultNumOfImageChannels = 3;

const string DataMaker::c_labelsFileName = "labels.txt";
const string DataMaker::c_dataInfoFileName = "data info.txt";

mutex DataMaker::s_outputMutex;

bool DataMaker::ParseArguments(int argc, char *argv[])
{
	if (!ParseArgument(argc, argv, c_inputFolderSignature, m_inputFolder) &&
		!ParseArgument(argc, argv, c_inputDataListSignature, m_inputDataListFile))
	{
		return false;
	}
	
	if (!ParseArgument(argc, argv, c_outputFolderSignature, m_outputFolder))
	{
		return false;
	}

	if (!ParseArgument(argc, argv, c_imageSizeSignature, m_imageSize))
	{
		m_imageSize = c_defaultImageSize;
	}

	if (!ParseArgument(argc, argv, c_numImageChannelsSignature, m_numImageChannels))
	{
		m_numImageChannels = c_defaultNumOfImageChannels;
	}
	
	return true;
}

void DataMaker::Initialize(DataMakerMode dataMakerMode)
{
	ShipAssert(_mkdir(m_outputFolder.c_str()) == 0 || errno == EEXIST, "Problem creating directory \"" + m_outputFolder + "\".");

	if (dataMakerMode == DataMakerMode::Training)
	{
		// Allocating memory for calculation of mean image.
		int coresPerGPU = Config::NUM_CPU_CORES / Config::NUM_GPUS;
		m_deviceMeanImageBufferLength = m_imageSize * m_imageSize * m_numImageChannels;
		int currentDevice;
		CudaAssert(hipGetDevice(&currentDevice));
		for (int i = 0; i < Config::NUM_CPU_CORES; ++i)
		{
			CudaAssert(hipSetDevice(i / coresPerGPU));
			m_deviceMeanImageBuffers.push_back(NULL);
			CudaAssert(hipMalloc<uint>(&m_deviceMeanImageBuffers[i], m_deviceMeanImageBufferLength * sizeof(uint)));
			m_meanImagesAppliedCounts.push_back(0);
		}
		CudaAssert(hipSetDevice(currentDevice));
	}
}

DataMaker::~DataMaker()
{
	for (size_t i = 0; i < m_deviceMeanImageBuffers.size(); ++i)
	{
		CudaAssert(hipFree(m_deviceMeanImageBuffers[i]));
	}
}

__global__ void AddImageToMeanKernel(uchar* deviceImageBuffer, uint* meanImageBuffer, uint imageBufferSize)
{
	for (uint i = blockIdx.x * blockDim.x + threadIdx.x; i < imageBufferSize; i += gridDim.x * blockDim.x)
	{
		meanImageBuffer[i] += deviceImageBuffer[i];
	}
}

void DataMaker::MakeDataPart(const vector<string>& data, string folder, int partIndex, DataMakerMode dataMakerMode)
{
	DebugAssert(partIndex < Config::NUM_CPU_CORES, "There is more data parts then CPU cores!");

	// Setting up the GPU.
	int coresPerGPU = Config::NUM_CPU_CORES / Config::NUM_GPUS;
	CudaAssert(hipSetDevice(partIndex / coresPerGPU));
	hipStream_t stream;
	CudaAssert(hipStreamCreate(&stream));
	nppSetStream(stream);

	// Setting up data positions.
	size_t dataPerCore = data.size() / Config::NUM_CPU_CORES;
	size_t startIndex = partIndex * dataPerCore;
	size_t endIndex = partIndex + 1 < Config::NUM_CPU_CORES ? (partIndex + 1) * dataPerCore : data.size();
	size_t dataToPrepareCnt = endIndex - startIndex;
	
	// Preparing data.
	DataParserFactory dataParserFactory;
	DataParser* dataParser;
	ImageData* image;
	uint doneCnt = 0;
	uint percentStep = (uint)max(dataPerCore / 20, (size_t)1);
	high_resolution_clock::time_point startTime = high_resolution_clock::now();
	for (size_t i = startIndex; i < endIndex; ++i)
	{
		// Finding file extension.
		string dataExtension = GetExtension(data[i]);
		if (dataExtension == "")
		{
			cout << "Encountered data without extension! File name: " << data[i] << endl;
			continue;
		}

		// Finding appropriate parser for that extension.
		dataParser = dataParserFactory.GetDataParser(dataExtension);

		// Parsing and preparing data.
		if (dataMakerMode == DataMakerMode::Training)
		{
			image = dataParser->LoadImage(m_inputFolder + "\\" + folder + "\\" + data[i]);
		}
		else
		{
			image = dataParser->LoadImage(data[i]);
		}
		ShipAssert(image->GetNumOfChannels() == m_numImageChannels, "Encountered image with invalid number of channels! File name: " + GetFileName(data[i]));

		if (dataMakerMode == DataMakerMode::Training)
		{
			HandleImageForTraining(image, data[i], dataParser, stream, partIndex, folder);
		}
		else if (dataMakerMode == DataMakerMode::Featurization)
		{
			HandleImageForFeaturization(image, GetFileName(data[i]), dataParser, stream);
		}
		else if (dataMakerMode == DataMakerMode::DatasetExtension)
		{
			HandleImageForDatasetExtension(image, GetFileName(data[i]), dataParser, stream);
		}

		delete image;

		if (++doneCnt % percentStep == 0)
		{
			high_resolution_clock::time_point endTime = high_resolution_clock::now();
			long long durationInSeconds = duration_cast<seconds>(endTime - startTime).count();
			long long numHours = durationInSeconds / 3600;
			long long numMinutes = (durationInSeconds - numHours * 3600) / 60;
			long long numSeconds = durationInSeconds - numHours * 3600 - numMinutes * 60;
			uint preparedPercent = (uint)(ceil(doneCnt * 100.0 / dataToPrepareCnt));

			lock_guard<mutex> lock(s_outputMutex);
			cout << "datamaker " << partIndex + 1 << " prepared: " << doneCnt << "/" << dataToPrepareCnt <<
				" (" << preparedPercent << "%) " << folder << " samples (Took: " << numHours << "h " << numMinutes << "min " << numSeconds << "s)" << endl;
		}
	}

	// Cleanup.
	CudaAssert(hipStreamDestroy(stream));
}

void DataMaker::HandleImageForTraining(ImageData* image, string imageName, DataParser* dataParser, hipStream_t stream, int partIndex, string folder)
{
	ImageData* resizedImage = dataParser->ResizeImageWithCropCu(*image, m_imageSize, m_imageSize, ResizeMode::ResizeToSmaller, CropMode::CropCentral, stream);
	int numBlocks = Config::MAX_NUM_FULL_BLOCKS;
	int numThreads = Config::MAX_NUM_THREADS;
	ShipAssert(m_deviceMeanImageBufferLength == resizedImage->GetBufferSize() / sizeof(uchar), "Resized image has unexpected buffer size!");
	if (m_deviceMeanImageBufferLength <= Config::MAX_NUM_THREADS)
	{
		numBlocks = 1;
		numThreads = m_deviceMeanImageBufferLength;
	}
	else if (m_deviceMeanImageBufferLength < Config::MAX_NUM_FULL_BLOCKS * Config::MAX_NUM_THREADS)
	{
		numThreads = RoundUp(DivideUp(m_deviceMeanImageBufferLength, Config::MAX_NUM_FULL_BLOCKS), Config::WARP_SIZE);
	}
	LAUNCH_KERNEL_ASYNC(AddImageToMeanKernel, numBlocks, numThreads, stream)(resizedImage->m_deviceImageBuffer, m_deviceMeanImageBuffers[partIndex],
		m_deviceMeanImageBufferLength);
	++m_meanImagesAppliedCounts[partIndex];
	
	dataParser->SaveImage(*resizedImage, m_outputFolder + "\\" + folder + "\\" + imageName);
	delete resizedImage;
}

void DataMaker::HandleImageForFeaturization(ImageData* image, string imageName, DataParser* dataParser, hipStream_t stream)
{
	ImageData* resizedImage = dataParser->ResizeImageWithCropCu(*image, m_imageSize, m_imageSize, ResizeMode::ResizeToSmaller, CropMode::CropCentral, stream);
	dataParser->SaveImage(*resizedImage, m_outputFolder + "\\" + imageName);
	delete resizedImage;
}

void DataMaker::HandleImageForDatasetExtension(ImageData* image, string imageName, DataParser* dataParser, hipStream_t stream)
{
	// TODO: try this with cuda crop

	// Make image square patch.
	ImageData* imagePatch;
	if (image->GetWidth() == image->GetHeight())
	{
		imagePatch = image;
	}
	else if (image->GetWidth() > image->GetHeight())
	{
		uint cropHeight = image->GetHeight();
		uint cropWidth = cropHeight;
		uint cropPositionX = (image->GetWidth() - cropWidth) / 2;
		uint cropPositionY = 0;
		imagePatch = dataParser->CropImage(*image, cropPositionX, cropPositionY, cropWidth, cropHeight, false);
	}
	else
	{
		uint cropWidth = image->GetWidth();
		uint cropHeight = cropWidth;
		uint cropPositionX = 0;
		uint cropPositionY = (image->GetHeight() - cropHeight) / 2;
		imagePatch = dataParser->CropImage(*image, cropPositionX, cropPositionY, cropWidth, cropHeight, false);
	}

	// Save the patch itself.
	dataParser->SaveImage(*imagePatch, m_outputFolder + "\\" + GetFileNameWithoutExtension(imageName) + "_0." + GetExtension(imageName));

	// Save the patches flip.
	ImageData* imagePatchFlip = dataParser->CropImage(*imagePatch, 0, 0, imagePatch->GetWidth(), imagePatch->GetHeight(), true);
	dataParser->SaveImage(*imagePatchFlip, m_outputFolder + "\\" + GetFileNameWithoutExtension(imageName) + "_1." + GetExtension(imageName));
	delete imagePatchFlip;

	// Save the crops and their flips.
	const float c_patchCropPercentage = 0.875;
	uint patchCropSize = (uint)(c_patchCropPercentage * imagePatch->GetWidth());
	float patchCropMove = (float)(imagePatch->GetWidth() - patchCropSize) / 2.0;
	for (int i = 0; i <= 2; ++i)
	{
		for (int j = 0; j <= 2; ++j)
		{
			uint patchCropPositionX = (uint)(i * patchCropMove);
			uint patchCropPositionY = (uint)(j * patchCropMove);

			// Save the crop.
			ImageData* patchCrop = dataParser->CropImage(*imagePatch, patchCropPositionX, patchCropPositionY, patchCropSize, patchCropSize, false);
			dataParser->SaveImage(*patchCrop, m_outputFolder + "\\" + GetFileNameWithoutExtension(imageName) + "_" + to_string(2 * (i * 3 + j + 1)) + "." + GetExtension(imageName));
			delete patchCrop;

			// Save the crop flip.
			ImageData* patchCropFlip = dataParser->CropImage(*imagePatch, patchCropPositionX, patchCropPositionY, patchCropSize, patchCropSize, true);
			dataParser->SaveImage(*patchCropFlip, m_outputFolder + "\\" + GetFileNameWithoutExtension(imageName) + "_" + to_string(2 * (i * 3 + j + 1) + 1) + "." + GetExtension(imageName));
			delete patchCropFlip;
		}
	}

	if (imagePatch != image)
	{
		delete imagePatch;
	}
}

void DataMaker::MakeData(string folder, DataMakerMode dataMakerMode)
{
	vector<string> images;
	if (dataMakerMode == DataMakerMode::Training)
	{
		// Creating output folder.
		string outputFolder = m_outputFolder + "\\" + folder;
		ShipAssert(_mkdir(outputFolder.c_str()) == 0 || errno == EEXIST, "Problem creating directory \"" + outputFolder + "\".");

		// Labels file will be copied to output location.
		ifstream labelsFile(m_inputFolder + "\\" + folder + "\\" + c_labelsFileName);
		ofstream dest(outputFolder + "\\" + c_labelsFileName);
		string imageName;
		int label;
		while (labelsFile >> imageName >> label)
		{
			images.push_back(imageName);

			// Writing label info.
			dest << imageName << " " << label << endl;
		}
		dest.close();
		labelsFile.close();
	}
	else
	{
		ifstream datalistFile(m_inputDataListFile);
		string imagePath;
		while (getline(datalistFile, imagePath))
		{
			images.push_back(imagePath);
		}
		datalistFile.close();
	}

	vector<thread> datamakerThreads;
#ifdef NDEBUG	
	int numThreads = Config::NUM_CPU_CORES;
#else
	int numThreads = 1;
#endif

	for (int i = 0; i < numThreads; ++i)
	{
		// TODO: find out why this doesn't work?  datamakerThreads.push_back(thread(&DataMaker::MakeDataPart, this, images, folder, i));
		datamakerThreads.push_back(thread([this, images, folder, i, dataMakerMode] { this->MakeDataPart(images, folder, i, dataMakerMode); }));
	}
	for (int i = 0; i < numThreads; ++i)
	{
		datamakerThreads[i].join();
	}

	cout << folder << " data prepared..." << endl << endl;
}

void DataMaker::MakeDataInfo()
{
	// Allocating temporary buffer to hold aggregated data from device mean image buffers.
	uint* tempImageBuffer;
	size_t tempImageBufferLength = m_deviceMeanImageBufferLength * sizeof(uint);
	CudaAssert(hipHostMalloc<uint>(&tempImageBuffer, tempImageBufferLength));

	// Aggregating device mean image buffers and calculating mean pixels.
	vector<unsigned long long> meanPixelValues;
	for (size_t i = 0; i < m_numImageChannels; ++i)
	{
		meanPixelValues.push_back(0);
	}
	for (size_t i = 0; i < m_deviceMeanImageBuffers.size(); ++i)
	{
		CudaAssert(hipMemcpy(tempImageBuffer, m_deviceMeanImageBuffers[i], tempImageBufferLength, hipMemcpyDeviceToHost));
		for (size_t j = 0; j < m_deviceMeanImageBufferLength; ++j)
		{
			meanPixelValues[j % m_numImageChannels] += tempImageBuffer[j];
		}
	}
	CudaAssert(hipHostFree(tempImageBuffer));
	unsigned long long meanImagesAppliedCount = 0;
	for (size_t i = 0; i < m_meanImagesAppliedCounts.size(); ++i)
	{
		meanImagesAppliedCount += m_meanImagesAppliedCounts[i];
	}
	unsigned long long pixelsPerChannelApplied = m_imageSize * m_imageSize * meanImagesAppliedCount;
	for (size_t i = 0; i < m_numImageChannels; ++i)
	{
		meanPixelValues[i] /= pixelsPerChannelApplied;
		ShipAssert(meanPixelValues[i] <= 255, "Calculated incorrect mean value! Value: " + to_string(meanPixelValues[i]));
	}

	// Writing data info file.
	ofstream dest(m_outputFolder + "\\" + c_dataInfoFileName);
	dest << "Mean pixel values: " << meanPixelValues[0];
	for (size_t i = 1; i < m_numImageChannels; ++i)
	{
		dest << ", " << meanPixelValues[i];
	}
	dest << endl;
}

void DataMaker::MakeDataForTraining()
{
	cout << endl;

	DataMakerMode dataMakerMode = DataMakerMode::Training;
	
	high_resolution_clock::time_point startTime = high_resolution_clock::now();
	Initialize(dataMakerMode);
	MakeData("train", dataMakerMode);
	MakeData("test", dataMakerMode);
	MakeDataInfo();
	high_resolution_clock::time_point endTime = high_resolution_clock::now();

	PrintTiming("Data is prepared for training", startTime, endTime);
}

void DataMaker::MakeDataForFeaturization()
{
	cout << endl;

	DataMakerMode dataMakerMode = DataMakerMode::Featurization;

	high_resolution_clock::time_point startTime = high_resolution_clock::now();
	Initialize(dataMakerMode);
	MakeData("", dataMakerMode);
	high_resolution_clock::time_point endTime = high_resolution_clock::now();

	PrintTiming("Data is prepared for featurization", startTime, endTime);
}

void DataMaker::MakeExtendedDataset()
{
	cout << endl;

	DataMakerMode dataMakerMode = DataMakerMode::DatasetExtension;

	high_resolution_clock::time_point startTime = high_resolution_clock::now();
	Initialize(dataMakerMode);
	MakeData("", dataMakerMode);
	high_resolution_clock::time_point endTime = high_resolution_clock::now();

	PrintTiming("Dataset is extended", startTime, endTime);
}

void DataMaker::PrintTiming(string operationMessage, high_resolution_clock::time_point operationStartTime,
	high_resolution_clock::time_point operationEndTime)
{
	long long durationInSeconds = duration_cast<seconds>(operationEndTime - operationStartTime).count();
	long long numHours = durationInSeconds / 3600;
	long long numMinutes = (durationInSeconds - numHours * 3600) / 60;
	long long numSeconds = durationInSeconds - numHours * 3600 - numMinutes * 60;
	cout << operationMessage << "! (Took: " << numHours << "h " << numMinutes << "min " << numSeconds << "s)" << endl;
}