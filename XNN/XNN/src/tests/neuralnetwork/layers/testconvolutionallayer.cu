#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------------------------------
// Copyrighted by Marko Rakita.
// Author: Marko Rakita
// File contains: Tests for convolutional layer.
// Created: 01/24/2016.
// ----------------------------------------------------------------------------------------------------

#include "include/testconvolutionallayer.cuh"

TestConvolutionalLayer::TestConvolutionalLayer(string outputFolder)
{
	m_outputFolder = outputFolder;

	// Registering tests.
	m_convolutionalLayerTests["doforwardprop"] = &TestConvolutionalLayer::TestDoForwardProp;
	m_convolutionalLayerTests["dobackwardprop"] = &TestConvolutionalLayer::TestDoBackwardProp;
}


bool TestConvolutionalLayer::HasTest(string testName)
{
	auto test = m_convolutionalLayerTests.find(testName);
	return test != m_convolutionalLayerTests.end();
}

void TestConvolutionalLayer::RunTest(string testName)
{
	auto test = m_convolutionalLayerTests.find(testName);
	TestingAssert(test != m_convolutionalLayerTests.end(), "Test not found!");

	((*this).*(test->second))();
}

void TestConvolutionalLayer::RunAllTests()
{
	for (auto test = m_convolutionalLayerTests.begin(); test != m_convolutionalLayerTests.end(); ++test)
	{
		((*this).*(test->second))();
		s_consoleHelper.SetConsoleForeground(ConsoleForeground::GREEN);
		cout << "Test " << test->first << " passed!" << endl << endl;
		s_consoleHelper.RevertConsoleForeground();
	}
}

//******************************************************************************************************
// Helper functions
//******************************************************************************************************

void TestConvolutionalLayer::TestDoForwardProp(uint inputNumChannels, uint inputDataWidth, uint inputDataHeight, uint inputDataCount, uint numFilters, uint filterWidth,
	uint filterHeight, int paddingX, int paddingY, uint stride)
{
	// Creating layers.
	MockInputLayer mockInputLayer(inputNumChannels, inputDataWidth, inputDataHeight, inputDataCount);
	float weightsDeviation = 0.01f;
	float biasesInitialValue = 1.0f;
	ActivationType activationType = ActivationType::ReLu;
	float filtersUpdateMomentum = 0.9f;
	float filtersUpdateDecay = 0.0005f;
	float filtersUpdateLearningRateProgressStep = 0.25f;
	float filtersUpdateStartingLearningRate = 0.01f;
	float filtersUpdateLearningRateUpdateFactor = 0.2f;
	float biasesUpdateMomentum = 0.9f;
	float biasesUpdateDecay = 0.f;
	float biasesUpdateLearningRateProgressStep = 0.5f;
	float biasesUpdateStartingLearningRate = 0.02f;
	float biasesUpdateLearningRateUpdateFactor = 0.1f;
	MockConvolutionalLayer mockConvolutionalLayer(inputNumChannels, inputDataWidth, inputDataHeight, inputDataCount, numFilters, filterWidth, filterHeight,
		inputNumChannels, weightsDeviation, biasesInitialValue, filtersUpdateMomentum, filtersUpdateDecay, filtersUpdateLearningRateProgressStep,
		filtersUpdateStartingLearningRate, filtersUpdateLearningRateUpdateFactor, biasesUpdateMomentum, biasesUpdateDecay, biasesUpdateLearningRateProgressStep,
		biasesUpdateStartingLearningRate, biasesUpdateLearningRateUpdateFactor, paddingX, paddingY, stride, activationType);
	mockConvolutionalLayer.AddPrevLayer(&mockInputLayer);
	ConvolutionalLayer convolutionalLayer(ParallelismMode::Data, 0, 0, 0, 1, inputNumChannels, inputDataWidth, inputDataHeight, inputDataCount, false, numFilters,
		filterWidth, filterHeight, inputNumChannels, false, weightsDeviation, false, biasesInitialValue, filtersUpdateMomentum, filtersUpdateDecay,
		filtersUpdateLearningRateProgressStep, filtersUpdateStartingLearningRate, filtersUpdateLearningRateUpdateFactor, biasesUpdateMomentum, biasesUpdateDecay,
		biasesUpdateLearningRateProgressStep, biasesUpdateStartingLearningRate, biasesUpdateLearningRateUpdateFactor, paddingX, paddingY, stride, activationType, false);
	convolutionalLayer.CopyFiltersFromHost(mockConvolutionalLayer.GetFiltersBuffer());
	convolutionalLayer.CopyBiasesFromHost(mockConvolutionalLayer.GetBiasesBuffer());
	convolutionalLayer.AddPrevLayer(&mockInputLayer);

	// Doing forward prop.
	PropagationMode propagationMode = PropagationMode::Train;
	mockInputLayer.LoadInputs();
	mockInputLayer.DoForwardProp(propagationMode);
	mockConvolutionalLayer.LoadInputs();
	convolutionalLayer.LoadInputs();
	convolutionalLayer.DoForwardProp(propagationMode);
	mockConvolutionalLayer.DoForwardProp(propagationMode);
	CudaAssert(hipDeviceSynchronize());
	
	// Transferring results to host.
	size_t activationsBufferSize = mockConvolutionalLayer.GetActivationBufferSize();
	float* convolutionalLayerActivationBuffer;
	CudaAssert(hipHostMalloc<float>(&convolutionalLayerActivationBuffer, activationsBufferSize));
	CudaAssert(hipMemcpy(convolutionalLayerActivationBuffer, convolutionalLayer.GetActivationDataBuffer(), activationsBufferSize, hipMemcpyDeviceToHost));
	
	// Checking correctness.
	bool correctResult = true;
	float firstDifference = 0.0f;
	bool foundDifferentFromZeroMock = false;
	bool foundDifferentFromZeroReg = false;
	size_t activationsBufferLength = activationsBufferSize / sizeof(float);
	const float* mockConvolutionalLayerActivationBuffer = mockConvolutionalLayer.GetActivationDataBuffer();
	for (size_t i = 0; i < activationsBufferLength; ++i)
	{
		float diff = fabs(mockConvolutionalLayerActivationBuffer[i] - convolutionalLayerActivationBuffer[i]);
		if (correctResult && (diff > 0.001f || (diff > 0.0001f && diff > 0.5f * max(abs(mockConvolutionalLayerActivationBuffer[i]), abs(convolutionalLayerActivationBuffer[i])))))
		{
			correctResult = false;
			firstDifference = mockConvolutionalLayerActivationBuffer[i] - convolutionalLayerActivationBuffer[i];
			PrintComputationInfo(i, inputNumChannels, inputDataWidth, inputDataHeight, inputDataCount, numFilters, filterWidth, filterHeight, paddingX, paddingY,
				stride, mockConvolutionalLayer.GetInputDataBuffer(), mockConvolutionalLayer.GetFiltersBuffer(), mockConvolutionalLayerActivationBuffer[i],
				convolutionalLayerActivationBuffer[i]);
		}
		if (mockConvolutionalLayerActivationBuffer[i] != 0.0f)
		{
			foundDifferentFromZeroMock = true;
		}
		if (convolutionalLayerActivationBuffer[i] != 0.0f)
		{
			foundDifferentFromZeroReg = true;
		}
	}

	CudaAssert(hipHostFree(convolutionalLayerActivationBuffer));

	TestingAssert(foundDifferentFromZeroMock, "All mock convolutional activations are zeros! Input num channels: " + to_string(inputNumChannels) + "; Input data count: " +
		to_string(inputDataCount) + "; Number of filters: " + to_string(numFilters));
	TestingAssert(foundDifferentFromZeroReg, "All convolutional activations are zeros! Input num channels: " + to_string(inputNumChannels) + "; Input data count: " +
		to_string(inputDataCount) + "; Number of filters: " + to_string(numFilters));
	TestingAssert(correctResult, "Incorrect forward prop! First difference: " + to_string(firstDifference) + "; Input num channels: " + to_string(inputNumChannels) +
		"; Input data count: " + to_string(inputDataCount) + "; Number of filters: " + to_string(numFilters));
	
	cout << "Forward prop passed. Input num channels: " << inputNumChannels << "; Input data count: " << inputDataCount << "; Number of filters: " << numFilters << endl;
}

void TestConvolutionalLayer::PrintComputationInfo(size_t activationDifferentPixelIndex, uint inputNumChannels, uint inputDataWidth, uint inputDataHeight, uint inputDataCount,
	uint numFilters, uint filterWidth, uint filterHeight, int paddingX, int paddingY, uint stride, float* inputDataBuffer, float* filtersBuffer,
	float differentActivationPixelMock, float differentActivationPixelRegular)
{
	size_t dataIndex = activationDifferentPixelIndex % inputDataCount;
	uint numPatchesX = 1 + (uint)ceil((double)(2 * paddingX + inputDataWidth - filterWidth) / stride);
	uint numPatchesY = 1 + (uint)ceil((double)(2 * paddingY + inputDataHeight - filterHeight) / stride);
	size_t filterIndex = activationDifferentPixelIndex / (inputDataCount * numPatchesX * numPatchesY);
	size_t patchIndex = (activationDifferentPixelIndex - filterIndex * inputDataCount * numPatchesX * numPatchesY) / inputDataCount;
	size_t patchIndexY = patchIndex / numPatchesX;
	size_t patchIndexX = patchIndex % numPatchesX;

	cout << "Data pixels in patch that causes bad computation:" << endl << endl;
	int dataStartX = -paddingX + (int)(patchIndexX * stride);
	int dataStartY = -paddingY + (int)(patchIndexY * stride);
	for (size_t channel = 0; channel < inputNumChannels; ++channel)
	{
		cout << "Channel " << channel << ":" << endl;
		for (int i = dataStartY; i < dataStartY + (int)filterHeight; ++i)
		{
			for (int j = dataStartX; j < dataStartX + (int)filterWidth; ++j)
			{
				if (i < 0 || i >= (int)inputDataHeight || j < 0 || j >= (int)inputDataWidth)
				{
					cout << 0 << " ";
				}
				else
				{
					cout << inputDataBuffer[dataIndex + channel * inputDataCount * inputDataWidth * inputDataHeight + (i * inputDataWidth + j) * inputDataCount] << " ";
				}
			}
			cout << endl;
		}
	}

	cout << endl << "Filter pixels:" << endl << endl;
	for (size_t channel = 0; channel < inputNumChannels; ++channel)
	{
		cout << "Channel " << channel << ":" << endl;
		for (int i = 0; i < (int)filterHeight; ++i)
		{
			for (int j = 0; j < (int)filterWidth; ++j)
			{
				cout << filtersBuffer[filterIndex + channel * numFilters * filterWidth * filterHeight + (i * filterWidth + j) * numFilters] << " ";
			}
			cout << endl;
		}
	}

	cout << endl << "Computated pixel in mock layer: " << differentActivationPixelMock << endl;
	cout << "Computated pixel in regular layer: " << differentActivationPixelRegular << endl;
}

void TestConvolutionalLayer::TestDoBackwardProp(uint inputNumChannels, uint inputDataWidth, uint inputDataHeight, uint inputDataCount, uint numFilters, uint filterWidth,
	uint filterHeight, int paddingX, int paddingY, uint stride)
{
	// Creating layers.
	float dataScale = inputNumChannels > 3 ? 0.001f : 1.f;
	MockInputLayer mockInputLayer(inputNumChannels, inputDataWidth, inputDataHeight, inputDataCount, dataScale);
	float weightsDeviation = 0.01f;
	float biasesInitialValue = 1.0f;
	ActivationType activationType = ActivationType::ReLu;
	float filtersUpdateMomentum = 0.9f;
	float filtersUpdateDecay = 0.0005f;
	float filtersUpdateLearningRateProgressStep = 0.25f;
	float filtersUpdateStartingLearningRate = 0.01f;
	float filtersUpdateLearningRateUpdateFactor = 0.2f;
	float biasesUpdateMomentum = 0.9f;
	float biasesUpdateDecay = 0.f;
	float biasesUpdateLearningRateProgressStep = 0.5f;
	float biasesUpdateStartingLearningRate = 0.02f;
	float biasesUpdateLearningRateUpdateFactor = 0.1f;
	MockConvolutionalLayer mockConvolutionalLayer(inputNumChannels, inputDataWidth, inputDataHeight, inputDataCount, numFilters, filterWidth, filterHeight,
		inputNumChannels, weightsDeviation, biasesInitialValue, filtersUpdateMomentum, filtersUpdateDecay, filtersUpdateLearningRateProgressStep,
		filtersUpdateStartingLearningRate, filtersUpdateLearningRateUpdateFactor, biasesUpdateMomentum, biasesUpdateDecay, biasesUpdateLearningRateProgressStep,
		biasesUpdateStartingLearningRate, biasesUpdateLearningRateUpdateFactor, paddingX, paddingY, stride, activationType);
	mockConvolutionalLayer.AddPrevLayer(&mockInputLayer);
	ConvolutionalLayer convolutionalLayer(ParallelismMode::Data, 0, 0, 0, 1, inputNumChannels, inputDataWidth, inputDataHeight, inputDataCount, false, numFilters,
		filterWidth, filterHeight, inputNumChannels, false, weightsDeviation, false, biasesInitialValue, filtersUpdateMomentum, filtersUpdateDecay,
		filtersUpdateLearningRateProgressStep, filtersUpdateStartingLearningRate, filtersUpdateLearningRateUpdateFactor, biasesUpdateMomentum, biasesUpdateDecay,
		biasesUpdateLearningRateProgressStep, biasesUpdateStartingLearningRate, biasesUpdateLearningRateUpdateFactor, paddingX, paddingY, stride, activationType, false);
	convolutionalLayer.CopyFiltersFromHost(mockConvolutionalLayer.GetFiltersBuffer());
	convolutionalLayer.CopyBiasesFromHost(mockConvolutionalLayer.GetBiasesBuffer());
	convolutionalLayer.AddPrevLayer(&mockInputLayer);
	MockOutputLayer mockOutputLayer(convolutionalLayer.GetActivationDataSize() * convolutionalLayer.GetActivationNumChannels(), inputDataCount, LossFunctionType::LogisticRegression, false, 0, true);
	mockConvolutionalLayer.AddNextLayer(&mockOutputLayer);
	convolutionalLayer.AddNextLayer(&mockOutputLayer);

	// Doing forward and backward prop.
	PropagationMode propagationMode = PropagationMode::Train;
	mockInputLayer.LoadInputs();
	mockInputLayer.DoForwardProp(propagationMode);
	mockConvolutionalLayer.LoadInputs();
	convolutionalLayer.LoadInputs();
	convolutionalLayer.DoForwardProp(propagationMode);
	mockConvolutionalLayer.DoForwardProp(propagationMode);
	mockOutputLayer.DoBackwardProp();
	convolutionalLayer.LoadActivationGradients();
	convolutionalLayer.DoBackwardProp();
	mockConvolutionalLayer.LoadActivationGradients();
	mockConvolutionalLayer.DoBackwardProp();
	CudaAssert(hipDeviceSynchronize());

	// Transferring input gradients results to host.
	size_t inputGradientsBufferSize = mockInputLayer.GetActivationBufferSize();
	float* convolutionalLayerInputGradientsBuffer;
	CudaAssert(hipHostMalloc<float>(&convolutionalLayerInputGradientsBuffer, inputGradientsBufferSize));
	CudaAssert(hipMemcpy(convolutionalLayerInputGradientsBuffer, convolutionalLayer.GetInputGradientsBuffer(), inputGradientsBufferSize, hipMemcpyDeviceToHost));

	// Checking input gradients correctness.
	bool correctResult = true;
	size_t numDifferences = 0;
	float firstDifference = 0.f;
	float firstDifferentMock = 0.f;
	float firstDifferentReg = 0.f;
	bool foundDifferentFromZeroMock = false;
	bool foundDifferentFromZeroReg = false;
	size_t inputGradientsBufferLength = inputGradientsBufferSize / sizeof(float);
	const float* mockConvolutionalLayerInputGradientsBuffer = mockConvolutionalLayer.GetInputGradientsBuffer();
	const float maxDiff = inputNumChannels > 3 ? 0.005f : 0.2f;
	const float maxDiffPercentage = 2.0f;
	const float maxDiffPercentageThreshold = 0.001f;
	CompareBuffers(convolutionalLayerInputGradientsBuffer, mockConvolutionalLayerInputGradientsBuffer, inputGradientsBufferLength, maxDiff, maxDiffPercentage,
		maxDiffPercentageThreshold, correctResult, numDifferences, firstDifference, firstDifferentMock, firstDifferentReg, foundDifferentFromZeroMock,
		foundDifferentFromZeroReg);

	CudaAssert(hipHostFree(convolutionalLayerInputGradientsBuffer));

	TestingAssert(foundDifferentFromZeroMock, "All mock convolutional input gradients are zeros! Input num channels: " + to_string(inputNumChannels) +
		"; Input data count: " + to_string(inputDataCount) + "; Number of filters: " + to_string(numFilters));
	TestingAssert(foundDifferentFromZeroReg, "All convolutional input gradients are zeros! Input num channels: " + to_string(inputNumChannels) +
		"; Input data count: " + to_string(inputDataCount) + "; Number of filters: " + to_string(numFilters));
	TestingAssert(correctResult, "Incorrect backward prop (input gradients)! Num differences: " + to_string(numDifferences) + "; First difference: " +
		to_string(firstDifference) + "; First different mock input gradient: " + to_string(firstDifferentMock) + "; First different regular input gradient: " +
		to_string(firstDifferentReg) + "; Input num channels: " + to_string(inputNumChannels) + "; Input data count: " + to_string(inputDataCount) +
		"; Number of filters: " + to_string(numFilters));

	// Transferring filters gradients results to host.
	size_t filtersGradientsBufferSize = mockConvolutionalLayer.GetFiltersBufferSize();
	float* convolutionalLayerFiltersGradientsBuffer;
	CudaAssert(hipHostMalloc<float>(&convolutionalLayerFiltersGradientsBuffer, filtersGradientsBufferSize));
	CudaAssert(hipMemcpy(convolutionalLayerFiltersGradientsBuffer, convolutionalLayer.GetFiltersGradientsBuffer(), filtersGradientsBufferSize, hipMemcpyDeviceToHost));

	// Checking filters gradients correctness.
	correctResult = true;
	foundDifferentFromZeroMock = false;
	foundDifferentFromZeroReg = false;
	size_t filtersGradientsBufferLength = filtersGradientsBufferSize / sizeof(float);
	const float* mockConvolutionalLayerFiltersGradientsBuffer = mockConvolutionalLayer.GetFiltersGradientsBuffer();
	const float maxDiffFG = inputNumChannels > 3 ? 0.007f : 0.3f;
	const float maxDiffPercentageFG = inputNumChannels > 3 ? 2.0f : 0.5f;
	const float maxDiffPercentageThresholdFG = 0.001f;
	CompareBuffers(convolutionalLayerFiltersGradientsBuffer, mockConvolutionalLayerFiltersGradientsBuffer, filtersGradientsBufferLength, maxDiffFG, maxDiffPercentageFG,
		maxDiffPercentageThresholdFG, correctResult, numDifferences, firstDifference, firstDifferentMock, firstDifferentReg, foundDifferentFromZeroMock,
		foundDifferentFromZeroReg);

	CudaAssert(hipHostFree(convolutionalLayerFiltersGradientsBuffer));

	TestingAssert(foundDifferentFromZeroMock, "All mock convolutional filters gradients are zeros! Input num channels: " + to_string(inputNumChannels) +
		"; Input data count: " + to_string(inputDataCount) + "; Number of filters: " + to_string(numFilters));
	TestingAssert(foundDifferentFromZeroReg, "All convolutional filters gradients are zeros! Input num channels: " + to_string(inputNumChannels) +
		"; Input data count: " + to_string(inputDataCount) + "; Number of filters: " + to_string(numFilters));
	TestingAssert(correctResult, "Incorrect backward prop (filters gradients)! Num differences: " + to_string(numDifferences) + "; First difference: " +
		to_string(firstDifference) + "; First different mock filters gradient: " + to_string(firstDifferentMock) + "; First different regular filters gradient: " +
		to_string(firstDifferentReg) + "; Input num channels: " + to_string(inputNumChannels) + "; Input data count: " + to_string(inputDataCount) +
		"; Number of filters: " + to_string(numFilters));

	// Transferring biases gradients results to host.
	size_t biasesGradientsBufferSize = mockConvolutionalLayer.GetBiasesBufferSize();
	float* convolutionalLayerBiasesGradientsBuffer;
	CudaAssert(hipHostMalloc<float>(&convolutionalLayerBiasesGradientsBuffer, biasesGradientsBufferSize));
	CudaAssert(hipMemcpy(convolutionalLayerBiasesGradientsBuffer, convolutionalLayer.GetBiasesGradientsBuffer(), biasesGradientsBufferSize, hipMemcpyDeviceToHost));

	// Checking biases gradients correctness.
	correctResult = true;
	foundDifferentFromZeroMock = false;
	foundDifferentFromZeroReg = false;
	size_t biasesGradientsBufferLength = biasesGradientsBufferSize / sizeof(float);
	const float* mockConvolutionalLayerBiasesGradientsBuffer = mockConvolutionalLayer.GetBiasesGradientsBuffer();
	const float maxDiffBG = inputNumChannels > 3 ? 0.007f : 0.2f;
	const float maxDiffPercentageBG = inputNumChannels > 3 ? 2.0f : 0.5f;
	const float maxDiffPercentageThresholdBG = 0.001f;
	CompareBuffers(convolutionalLayerBiasesGradientsBuffer, mockConvolutionalLayerBiasesGradientsBuffer, biasesGradientsBufferLength, maxDiffBG, maxDiffPercentageBG,
		maxDiffPercentageThresholdBG, correctResult, numDifferences, firstDifference, firstDifferentMock, firstDifferentReg, foundDifferentFromZeroMock,
		foundDifferentFromZeroReg);

	CudaAssert(hipHostFree(convolutionalLayerBiasesGradientsBuffer));

	TestingAssert(foundDifferentFromZeroMock, "All mock convolutional biases gradients are zeros! Input num channels: " + to_string(inputNumChannels) +
		"; Input data count: " + to_string(inputDataCount) + "; Number of filters: " + to_string(numFilters));
	TestingAssert(foundDifferentFromZeroReg, "All convolutional biases gradients are zeros! Input num channels: " + to_string(inputNumChannels) +
		"; Input data count: " + to_string(inputDataCount) + "; Number of filters: " + to_string(numFilters));
	TestingAssert(correctResult, "Incorrect backward prop (biases gradients)! Num differences: " + to_string(numDifferences) + "; First difference: " +
		to_string(firstDifference) + "; First different mock biases gradient: " + to_string(firstDifferentMock) + "; First different regular biases gradient: " +
		to_string(firstDifferentReg) + "; Input num channels: " + to_string(inputNumChannels) + "; Input data count: " + to_string(inputDataCount) +
		"; Number of filters: " + to_string(numFilters));

	// Updating parameters.
	float progress = 0.6f;
	convolutionalLayer.UpdateLayerParameters(progress);
	mockConvolutionalLayer.UpdateLayerParameters(progress);
	CudaAssert(hipDeviceSynchronize());

	// Transferring filters to host.
	size_t filtersBufferSize = mockConvolutionalLayer.GetFiltersBufferSize();
	float* convolutionalLayerFiltersBuffer;
	CudaAssert(hipHostMalloc<float>(&convolutionalLayerFiltersBuffer, filtersBufferSize));
	CudaAssert(hipMemcpy(convolutionalLayerFiltersBuffer, convolutionalLayer.GetFiltersBuffer(), filtersBufferSize, hipMemcpyDeviceToHost));

	// Checking filters correctness.
	correctResult = true;
	foundDifferentFromZeroMock = false;
	foundDifferentFromZeroReg = false;
	size_t filtersBufferLength = filtersBufferSize / sizeof(float);
	const float* mockConvolutionalLayerFiltersBuffer = mockConvolutionalLayer.GetFiltersBuffer();
	const float maxDiffF = 0.01f;
	const float maxDiffPercentageF = 0.1f;
	const float maxDiffPercentageThresholdF = 0.005f;
	CompareBuffers(convolutionalLayerFiltersBuffer, mockConvolutionalLayerFiltersBuffer, filtersBufferLength, maxDiffF, maxDiffPercentageF, maxDiffPercentageThresholdF,
		correctResult, numDifferences, firstDifference, firstDifferentMock, firstDifferentReg, foundDifferentFromZeroMock, foundDifferentFromZeroReg);

	CudaAssert(hipHostFree(convolutionalLayerFiltersBuffer));

	TestingAssert(foundDifferentFromZeroMock, "All mock convolutional filters are zeros! Input num channels: " + to_string(inputNumChannels) +
		"; Input data count: " + to_string(inputDataCount) + "; Number of filters: " + to_string(numFilters));
	TestingAssert(foundDifferentFromZeroReg, "All convolutional filters are zeros! Input num channels: " + to_string(inputNumChannels) +
		"; Input data count: " + to_string(inputDataCount) + "; Number of filters: " + to_string(numFilters));
	TestingAssert(correctResult, "Incorrect backward prop (updated filters)! Num differences: " + to_string(numDifferences) + "; First difference: " +
		to_string(firstDifference) + "; First different mock filters: " + to_string(firstDifferentMock) + "; First different regular filters: " +
		to_string(firstDifferentReg) + "; Input num channels: " + to_string(inputNumChannels) + "; Input data count: " + to_string(inputDataCount) +
		"; Number of filters: " + to_string(numFilters));

	// Transferring biases to host.
	size_t biasesBufferSize = mockConvolutionalLayer.GetBiasesBufferSize();
	float* convolutionalLayerBiasesBuffer;
	CudaAssert(hipHostMalloc<float>(&convolutionalLayerBiasesBuffer, biasesBufferSize));
	CudaAssert(hipMemcpy(convolutionalLayerBiasesBuffer, convolutionalLayer.GetBiasesBuffer(), biasesBufferSize, hipMemcpyDeviceToHost));

	// Checking biases correctness.
	correctResult = true;
	foundDifferentFromZeroMock = false;
	foundDifferentFromZeroReg = false;
	size_t biasesBufferLength = biasesBufferSize / sizeof(float);
	const float* mockConvolutionalLayerBiasesBuffer = mockConvolutionalLayer.GetBiasesBuffer();
	const float maxDiffB = 0.01f;
	const float maxDiffPercentageB = 0.1f;
	const float maxDiffPercentageThresholdB = 0.005f;
	CompareBuffers(convolutionalLayerBiasesBuffer, mockConvolutionalLayerBiasesBuffer, biasesBufferLength, maxDiffB, maxDiffPercentageB, maxDiffPercentageThresholdB,
		correctResult, numDifferences, firstDifference, firstDifferentMock, firstDifferentReg, foundDifferentFromZeroMock, foundDifferentFromZeroReg);

	CudaAssert(hipHostFree(convolutionalLayerBiasesBuffer));

	TestingAssert(foundDifferentFromZeroMock, "All mock convolutional biases are zeros! Input num channels: " + to_string(inputNumChannels) +
		"; Input data count: " + to_string(inputDataCount) + "; Number of filters: " + to_string(numFilters));
	TestingAssert(foundDifferentFromZeroReg, "All convolutional biases are zeros! Input num channels: " + to_string(inputNumChannels) +
		"; Input data count: " + to_string(inputDataCount) + "; Number of filters: " + to_string(numFilters));
	TestingAssert(correctResult, "Incorrect backward prop (updated biases)! Num differences: " + to_string(numDifferences) + "; First difference: " +
		to_string(firstDifference) + "; First different mock biases: " + to_string(firstDifferentMock) + "; First different regular biases: " +
		to_string(firstDifferentReg) + "; Input num channels: " + to_string(inputNumChannels) + "; Input data count: " + to_string(inputDataCount) +
		"; Number of filters: " + to_string(numFilters));

	cout << "Backward prop passed. Input num channels: " << inputNumChannels << "; Input data count: " << inputDataCount << "; Number of filters: " << numFilters << endl;
}

//******************************************************************************************************
// Tests
//******************************************************************************************************

void TestConvolutionalLayer::TestDoForwardProp()
{
	// lastBatch == true

	// inputNumChannels == 3
	TestDoForwardProp(3 /*inputNumChannels*/, 224 /*inputDataWidth*/, 224 /*inputDataHeight*/, 97 /*inputDataCount*/, 64 /*numFilters*/, 11 /*filterWidth*/,
		11 /*filterHeight*/, 0 /*paddingX*/, 0 /*paddingY*/, 4 /*stride*/);
	TestDoForwardProp(3 /*inputNumChannels*/, 224 /*inputDataWidth*/, 224 /*inputDataHeight*/, 35 /*inputDataCount*/, 48 /*numFilters*/, 11 /*filterWidth*/,
		11 /*filterHeight*/, 0 /*paddingX*/, 0 /*paddingY*/, 4 /*stride*/);
	TestDoForwardProp(3 /*inputNumChannels*/, 224 /*inputDataWidth*/, 224 /*inputDataHeight*/, 1 /*inputDataCount*/, 32 /*numFilters*/, 11 /*filterWidth*/,
		11 /*filterHeight*/, 1 /*paddingX*/, 1 /*paddingY*/, 4 /*stride*/);
	TestDoForwardProp(3 /*inputNumChannels*/, 224 /*inputDataWidth*/, 224 /*inputDataHeight*/, 127 /*inputDataCount*/, 16 /*numFilters*/, 11 /*filterWidth*/,
		11 /*filterHeight*/, 1 /*paddingX*/, 1 /*paddingY*/, 4 /*stride*/);

	// inputNumChannels % 4 == 0
	TestDoForwardProp(20 /*inputNumChannels*/, 13 /*inputDataWidth*/, 13 /*inputDataHeight*/, 27 /*inputDataCount*/, 384 /*numFilters*/, 3 /*filterWidth*/,
		3 /*filterHeight*/, 1 /*paddingX*/, 1 /*paddingY*/, 1 /*stride*/);
	TestDoForwardProp(20 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 111 /*inputDataCount*/, 192 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);
	TestDoForwardProp(44 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 86 /*inputDataCount*/, 32 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);
	TestDoForwardProp(44 /*inputNumChannels*/, 55 /*inputDataWidth*/, 55 /*inputDataHeight*/, 99 /*inputDataCount*/, 16 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);

	// inputNumChannels % 8 == 0
	TestDoForwardProp(32 /*inputNumChannels*/, 13 /*inputDataWidth*/, 13 /*inputDataHeight*/, 22 /*inputDataCount*/, 256 /*numFilters*/, 3 /*filterWidth*/,
		3 /*filterHeight*/, 1 /*paddingX*/, 1 /*paddingY*/, 1 /*stride*/);
	TestDoForwardProp(48 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 79 /*inputDataCount*/, 64 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);
	TestDoForwardProp(64 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 88 /*inputDataCount*/, 32 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);
	TestDoForwardProp(128 /*inputNumChannels*/, 55 /*inputDataWidth*/, 55 /*inputDataHeight*/, 125 /*inputDataCount*/, 16 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);

	
	// lastBatch == false

	// inputNumChannels == 3

	// inputDataCount % 128 == 0
	TestDoForwardProp(3 /*inputNumChannels*/, 224 /*inputDataWidth*/, 224 /*inputDataHeight*/, 128 /*inputDataCount*/, 64 /*numFilters*/, 11 /*filterWidth*/,
		11 /*filterHeight*/, 0 /*paddingX*/, 0 /*paddingY*/, 4 /*stride*/);
	TestDoForwardProp(3 /*inputNumChannels*/, 224 /*inputDataWidth*/, 224 /*inputDataHeight*/, 128 /*inputDataCount*/, 48 /*numFilters*/, 11 /*filterWidth*/,
		11 /*filterHeight*/, 0 /*paddingX*/, 0 /*paddingY*/, 4 /*stride*/);
	TestDoForwardProp(3 /*inputNumChannels*/, 224 /*inputDataWidth*/, 224 /*inputDataHeight*/, 128 /*inputDataCount*/, 32 /*numFilters*/, 11 /*filterWidth*/,
		11 /*filterHeight*/, 1 /*paddingX*/, 1 /*paddingY*/, 4 /*stride*/);
	TestDoForwardProp(3 /*inputNumChannels*/, 224 /*inputDataWidth*/, 224 /*inputDataHeight*/, 128 /*inputDataCount*/, 16 /*numFilters*/, 11 /*filterWidth*/,
		11 /*filterHeight*/, 1 /*paddingX*/, 1 /*paddingY*/, 4 /*stride*/);

	// inputDataCount % 64 == 0
	TestDoForwardProp(3 /*inputNumChannels*/, 224 /*inputDataWidth*/, 224 /*inputDataHeight*/, 64 /*inputDataCount*/, 64 /*numFilters*/, 11 /*filterWidth*/,
		11 /*filterHeight*/, 0 /*paddingX*/, 0 /*paddingY*/, 4 /*stride*/);
	TestDoForwardProp(3 /*inputNumChannels*/, 224 /*inputDataWidth*/, 224 /*inputDataHeight*/, 64 /*inputDataCount*/, 48 /*numFilters*/, 11 /*filterWidth*/,
		11 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 4 /*stride*/);
	TestDoForwardProp(3 /*inputNumChannels*/, 224 /*inputDataWidth*/, 224 /*inputDataHeight*/, 64 /*inputDataCount*/, 32 /*numFilters*/, 11 /*filterWidth*/,
		11 /*filterHeight*/, 0 /*paddingX*/, 0 /*paddingY*/, 4 /*stride*/);
	TestDoForwardProp(3 /*inputNumChannels*/, 224 /*inputDataWidth*/, 224 /*inputDataHeight*/, 64 /*inputDataCount*/, 16 /*numFilters*/, 11 /*filterWidth*/,
		11 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 4 /*stride*/);

	// inputDataCount % 32 == 0
	TestDoForwardProp(3 /*inputNumChannels*/, 224 /*inputDataWidth*/, 224 /*inputDataHeight*/, 32 /*inputDataCount*/, 64 /*numFilters*/, 11 /*filterWidth*/,
		11 /*filterHeight*/, 0 /*paddingX*/, 0 /*paddingY*/, 4 /*stride*/);
	TestDoForwardProp(3 /*inputNumChannels*/, 224 /*inputDataWidth*/, 224 /*inputDataHeight*/, 32 /*inputDataCount*/, 48 /*numFilters*/, 11 /*filterWidth*/,
		11 /*filterHeight*/, 1 /*paddingX*/, 1 /*paddingY*/, 4 /*stride*/);
	TestDoForwardProp(3 /*inputNumChannels*/, 224 /*inputDataWidth*/, 224 /*inputDataHeight*/, 32 /*inputDataCount*/, 32 /*numFilters*/, 11 /*filterWidth*/,
		11 /*filterHeight*/, 0 /*paddingX*/, 0 /*paddingY*/, 4 /*stride*/);
	TestDoForwardProp(3 /*inputNumChannels*/, 224 /*inputDataWidth*/, 224 /*inputDataHeight*/, 32 /*inputDataCount*/, 16 /*numFilters*/, 11 /*filterWidth*/,
		11 /*filterHeight*/, 1 /*paddingX*/, 1 /*paddingY*/, 4 /*stride*/);

	// inputNumChannels % 4 == 0

	// inputDataCount % 128 == 0
	TestDoForwardProp(20 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 128 /*inputDataCount*/, 128 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);
	TestDoForwardProp(20 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 128 /*inputDataCount*/, 64 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);
	TestDoForwardProp(44 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 128 /*inputDataCount*/, 32 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);
	TestDoForwardProp(44 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 128 /*inputDataCount*/, 16 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);

	// inputDataCount % 64 == 0
	TestDoForwardProp(20 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 64 /*inputDataCount*/, 128 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);
	TestDoForwardProp(20 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 64 /*inputDataCount*/, 64 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);
	TestDoForwardProp(44 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 64 /*inputDataCount*/, 32 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);
	TestDoForwardProp(44 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 64 /*inputDataCount*/, 16 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);

	// inputDataCount % 32 == 0
	TestDoForwardProp(20 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 32 /*inputDataCount*/, 128 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);
	TestDoForwardProp(20 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 32 /*inputDataCount*/, 64 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);
	TestDoForwardProp(44 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 32 /*inputDataCount*/, 32 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);
	TestDoForwardProp(44 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 32 /*inputDataCount*/, 16 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);

	// inputNumChannels % 8 == 0

	// inputDataCount % 128 == 0
	TestDoForwardProp(32 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 128 /*inputDataCount*/, 128 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 1 /*paddingX*/, 1 /*paddingY*/, 1 /*stride*/);
	TestDoForwardProp(48 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 128 /*inputDataCount*/, 64 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);
	TestDoForwardProp(64 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 128 /*inputDataCount*/, 32 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 1 /*paddingX*/, 1 /*paddingY*/, 1 /*stride*/);
	TestDoForwardProp(128 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 128 /*inputDataCount*/, 16 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);

	// inputDataCount % 64 == 0
	TestDoForwardProp(32 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 64 /*inputDataCount*/, 128 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);
	TestDoForwardProp(48 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 64 /*inputDataCount*/, 64 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);
	TestDoForwardProp(64 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 64 /*inputDataCount*/, 32 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 1 /*paddingX*/, 1 /*paddingY*/, 1 /*stride*/);
	TestDoForwardProp(128 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 64 /*inputDataCount*/, 16 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 0 /*paddingX*/, 0 /*paddingY*/, 1 /*stride*/);

	// inputDataCount % 32 == 0
	TestDoForwardProp(32 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 32 /*inputDataCount*/, 128 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 0 /*paddingX*/, 0 /*paddingY*/, 1 /*stride*/);
	TestDoForwardProp(48 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 32 /*inputDataCount*/, 64 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 1 /*paddingX*/, 1 /*paddingY*/, 1 /*stride*/);
	TestDoForwardProp(64 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 32 /*inputDataCount*/, 32 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);
	TestDoForwardProp(128 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 32 /*inputDataCount*/, 16 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);
}

void TestConvolutionalLayer::TestDoBackwardProp()
{
	// lastBatch == true

	// inputNumChannels == 3

	TestDoBackwardProp(3 /*inputNumChannels*/, 224 /*inputDataWidth*/, 224 /*inputDataHeight*/, 97 /*inputDataCount*/, 64 /*numFilters*/, 11 /*filterWidth*/,
		11 /*filterHeight*/, 0 /*paddingX*/, 0 /*paddingY*/, 4 /*stride*/);
	TestDoBackwardProp(3 /*inputNumChannels*/, 224 /*inputDataWidth*/, 224 /*inputDataHeight*/, 35 /*inputDataCount*/, 48 /*numFilters*/, 11 /*filterWidth*/,
		11 /*filterHeight*/, 0 /*paddingX*/, 0 /*paddingY*/, 4 /*stride*/);
	TestDoBackwardProp(3 /*inputNumChannels*/, 224 /*inputDataWidth*/, 224 /*inputDataHeight*/, 1 /*inputDataCount*/, 32 /*numFilters*/, 11 /*filterWidth*/,
		11 /*filterHeight*/, 1 /*paddingX*/, 1 /*paddingY*/, 4 /*stride*/);
	TestDoBackwardProp(3 /*inputNumChannels*/, 224 /*inputDataWidth*/, 224 /*inputDataHeight*/, 127 /*inputDataCount*/, 16 /*numFilters*/, 11 /*filterWidth*/,
		11 /*filterHeight*/, 1 /*paddingX*/, 1 /*paddingY*/, 4 /*stride*/);

	// inputNumChannels % 4 == 0
	// TODO: Currently unsupported, uncomment here and below if you support this one day.
	//TestDoBackwardProp(20 /*inputNumChannels*/, 13 /*inputDataWidth*/, 13 /*inputDataHeight*/, 27 /*inputDataCount*/, 384 /*numFilters*/, 3 /*filterWidth*/,
	//	3 /*filterHeight*/, 1 /*paddingX*/, 1 /*paddingY*/, 1 /*stride*/);
	//TestDoBackwardProp(20 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 111 /*inputDataCount*/, 192 /*numFilters*/, 5 /*filterWidth*/,
	//	5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);
	//TestDoBackwardProp(44 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 86 /*inputDataCount*/, 32 /*numFilters*/, 5 /*filterWidth*/,
	//	5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);
	//TestDoBackwardProp(44 /*inputNumChannels*/, 55 /*inputDataWidth*/, 55 /*inputDataHeight*/, 99 /*inputDataCount*/, 16 /*numFilters*/, 5 /*filterWidth*/,
	//	5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);

	// inputNumChannels % 8 == 0
	TestDoBackwardProp(32 /*inputNumChannels*/, 13 /*inputDataWidth*/, 13 /*inputDataHeight*/, 22 /*inputDataCount*/, 256 /*numFilters*/, 3 /*filterWidth*/,
		3 /*filterHeight*/, 1 /*paddingX*/, 1 /*paddingY*/, 1 /*stride*/);
	TestDoBackwardProp(48 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 79 /*inputDataCount*/, 64 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);
	TestDoBackwardProp(64 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 88 /*inputDataCount*/, 32 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);
	TestDoBackwardProp(128 /*inputNumChannels*/, 55 /*inputDataWidth*/, 55 /*inputDataHeight*/, 125 /*inputDataCount*/, 16 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);


	// lastBatch == false

	// inputNumChannels == 3

	// inputDataCount % 128 == 0
	TestDoBackwardProp(3 /*inputNumChannels*/, 224 /*inputDataWidth*/, 224 /*inputDataHeight*/, 128 /*inputDataCount*/, 64 /*numFilters*/, 11 /*filterWidth*/,
		11 /*filterHeight*/, 0 /*paddingX*/, 0 /*paddingY*/, 4 /*stride*/);
	TestDoBackwardProp(3 /*inputNumChannels*/, 224 /*inputDataWidth*/, 224 /*inputDataHeight*/, 128 /*inputDataCount*/, 48 /*numFilters*/, 11 /*filterWidth*/,
		11 /*filterHeight*/, 0 /*paddingX*/, 0 /*paddingY*/, 4 /*stride*/);
	TestDoBackwardProp(3 /*inputNumChannels*/, 224 /*inputDataWidth*/, 224 /*inputDataHeight*/, 128 /*inputDataCount*/, 32 /*numFilters*/, 11 /*filterWidth*/,
		11 /*filterHeight*/, 1 /*paddingX*/, 1 /*paddingY*/, 4 /*stride*/);
	TestDoBackwardProp(3 /*inputNumChannels*/, 224 /*inputDataWidth*/, 224 /*inputDataHeight*/, 128 /*inputDataCount*/, 16 /*numFilters*/, 11 /*filterWidth*/,
		11 /*filterHeight*/, 1 /*paddingX*/, 1 /*paddingY*/, 4 /*stride*/);

	// inputDataCount % 64 == 0
	TestDoBackwardProp(3 /*inputNumChannels*/, 224 /*inputDataWidth*/, 224 /*inputDataHeight*/, 64 /*inputDataCount*/, 64 /*numFilters*/, 11 /*filterWidth*/,
		11 /*filterHeight*/, 1 /*paddingX*/, 1 /*paddingY*/, 4 /*stride*/);
	TestDoBackwardProp(3 /*inputNumChannels*/, 224 /*inputDataWidth*/, 224 /*inputDataHeight*/, 64 /*inputDataCount*/, 48 /*numFilters*/, 11 /*filterWidth*/,
		11 /*filterHeight*/, 1 /*paddingX*/, 1 /*paddingY*/, 4 /*stride*/);
	TestDoBackwardProp(3 /*inputNumChannels*/, 224 /*inputDataWidth*/, 224 /*inputDataHeight*/, 64 /*inputDataCount*/, 32 /*numFilters*/, 11 /*filterWidth*/,
		11 /*filterHeight*/, 0 /*paddingX*/, 0 /*paddingY*/, 4 /*stride*/);
	TestDoBackwardProp(3 /*inputNumChannels*/, 224 /*inputDataWidth*/, 224 /*inputDataHeight*/, 64 /*inputDataCount*/, 16 /*numFilters*/, 11 /*filterWidth*/,
		11 /*filterHeight*/, 0 /*paddingX*/, 0 /*paddingY*/, 4 /*stride*/);

	// inputDataCount % 32 == 0
	TestDoBackwardProp(3 /*inputNumChannels*/, 224 /*inputDataWidth*/, 224 /*inputDataHeight*/, 32 /*inputDataCount*/, 64 /*numFilters*/, 11 /*filterWidth*/,
		11 /*filterHeight*/, 0 /*paddingX*/, 0 /*paddingY*/, 4 /*stride*/);
	TestDoBackwardProp(3 /*inputNumChannels*/, 224 /*inputDataWidth*/, 224 /*inputDataHeight*/, 32 /*inputDataCount*/, 48 /*numFilters*/, 11 /*filterWidth*/,
		11 /*filterHeight*/, 1 /*paddingX*/, 1 /*paddingY*/, 4 /*stride*/);
	TestDoBackwardProp(3 /*inputNumChannels*/, 224 /*inputDataWidth*/, 224 /*inputDataHeight*/, 32 /*inputDataCount*/, 32 /*numFilters*/, 11 /*filterWidth*/,
		11 /*filterHeight*/, 1 /*paddingX*/, 1 /*paddingY*/, 4 /*stride*/);
	TestDoBackwardProp(3 /*inputNumChannels*/, 224 /*inputDataWidth*/, 224 /*inputDataHeight*/, 32 /*inputDataCount*/, 16 /*numFilters*/, 11 /*filterWidth*/,
		11 /*filterHeight*/, 0 /*paddingX*/, 0 /*paddingY*/, 4 /*stride*/);

	// inputNumChannels % 4 == 0

	// inputDataCount % 128 == 0
	//TestDoBackwardProp(20 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 128 /*inputDataCount*/, 128 /*numFilters*/, 5 /*filterWidth*/,
	//	5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);
	//TestDoBackwardProp(20 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 128 /*inputDataCount*/, 64 /*numFilters*/, 5 /*filterWidth*/,
	//	5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);
	//TestDoBackwardProp(44 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 128 /*inputDataCount*/, 32 /*numFilters*/, 5 /*filterWidth*/,
	//	5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);
	//TestDoBackwardProp(44 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 128 /*inputDataCount*/, 16 /*numFilters*/, 5 /*filterWidth*/,
	//	5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);

	// inputDataCount % 64 == 0
	//TestDoBackwardProp(20 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 64 /*inputDataCount*/, 128 /*numFilters*/, 5 /*filterWidth*/,
	//	5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);
	//TestDoBackwardProp(20 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 64 /*inputDataCount*/, 64 /*numFilters*/, 5 /*filterWidth*/,
	//	5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);
	//TestDoBackwardProp(44 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 64 /*inputDataCount*/, 32 /*numFilters*/, 5 /*filterWidth*/,
	//	5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);
	//TestDoBackwardProp(44 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 64 /*inputDataCount*/, 16 /*numFilters*/, 5 /*filterWidth*/,
	//	5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);

	// inputDataCount % 32 == 0
	//TestDoBackwardProp(20 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 32 /*inputDataCount*/, 128 /*numFilters*/, 5 /*filterWidth*/,
	//	5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);
	//TestDoBackwardProp(20 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 32 /*inputDataCount*/, 64 /*numFilters*/, 5 /*filterWidth*/,
	//	5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);
	//TestDoBackwardProp(44 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 32 /*inputDataCount*/, 32 /*numFilters*/, 5 /*filterWidth*/,
	//	5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);
	//TestDoBackwardProp(44 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 32 /*inputDataCount*/, 16 /*numFilters*/, 5 /*filterWidth*/,
	//	5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);

	// inputNumChannels % 8 == 0

	// inputDataCount % 128 == 0
	TestDoBackwardProp(32 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 128 /*inputDataCount*/, 128 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 1 /*paddingX*/, 1 /*paddingY*/, 1 /*stride*/);
	TestDoBackwardProp(48 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 128 /*inputDataCount*/, 64 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 1 /*paddingX*/, 1 /*paddingY*/, 1 /*stride*/);
	TestDoBackwardProp(64 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 128 /*inputDataCount*/, 32 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);
	TestDoBackwardProp(128 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 128 /*inputDataCount*/, 16 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);

	// inputDataCount % 64 == 0
	TestDoBackwardProp(32 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 64 /*inputDataCount*/, 128 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 0 /*paddingX*/, 0 /*paddingY*/, 1 /*stride*/);
	TestDoBackwardProp(48 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 64 /*inputDataCount*/, 64 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 0 /*paddingX*/, 0 /*paddingY*/, 1 /*stride*/);
	TestDoBackwardProp(64 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 64 /*inputDataCount*/, 32 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);
	TestDoBackwardProp(128 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 64 /*inputDataCount*/, 16 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);

	// inputDataCount % 32 == 0
	TestDoBackwardProp(32 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 32 /*inputDataCount*/, 128 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 2 /*paddingX*/, 2 /*paddingY*/, 1 /*stride*/);
	TestDoBackwardProp(48 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 32 /*inputDataCount*/, 64 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 0 /*paddingX*/, 0 /*paddingY*/, 1 /*stride*/);
	TestDoBackwardProp(64 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 32 /*inputDataCount*/, 32 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 1 /*paddingX*/, 1 /*paddingY*/, 1 /*stride*/);
	TestDoBackwardProp(128 /*inputNumChannels*/, 27 /*inputDataWidth*/, 27 /*inputDataHeight*/, 32 /*inputDataCount*/, 16 /*numFilters*/, 5 /*filterWidth*/,
		5 /*filterHeight*/, 1 /*paddingX*/, 1 /*paddingY*/, 1 /*stride*/);
}